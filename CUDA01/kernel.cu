
#include "hip/hip_runtime.h"


#include <stdio.h>


__device__ __managed__ int ret[1000];
__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int main(void) {
	//int *um_a, *um_b, *um_c; // unified memory values
	int h_a, h_b, h_c; // host values
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	// allocate memory for unified values
	//cudaMallocManaged((void **)&um_a, size);
	//cudaMallocManaged((void **)&um_b, size);
	//cudaMallocManaged((void **)&um_c, size);

	// allocate memory for device values
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	h_a = 2;
	h_b = 7;
	//&um_a = 4;
	//&um_b = 6;

	// Copy inputs to device
	hipMemcpy(d_a, &h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);
	//add << <1, 1 >> >(um_a, um_b, um_c);

	// Copy result back to host
	hipMemcpy(&h_c, d_c, size, hipMemcpyDeviceToHost);

	//printf("unified memory = %d\n", um_c);
	printf("device memory: %d\n", h_c);

	// Cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);



	return 0;
}
